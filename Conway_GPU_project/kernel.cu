﻿#include "cudaHelperFunctions.cuh"
#include "ConwayTable.cpp"


int main()
{
	//Host side variables
	std::vector<char> v = {0,0,0,0,0,0,
						   0,1,1,0,0,0,
						   0,1,1,0,0,0,
						   0,0,0,1,1,0,
						   0,0,0,1,1,0,
						   0,0,0,0,0,0}; 
	int n = 6;

	
	//Device side variables
	char* grid = nullptr;

	char* neighGrid = nullptr;
	char* neighGrid2 = nullptr;

	// Allocating memory on the device 
	auto err = hipMalloc((void**)&grid, n * n * sizeof(char));
	if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
	err = hipMalloc((void**)&neighGrid, n * n * sizeof(char));
	if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
	err = hipMalloc((void**)&neighGrid2, n * n * sizeof(char));
	if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }

	// Copying data from the host to the device
	err = hipMemcpy(grid, v.data(), n * n * sizeof(char), hipMemcpyHostToDevice);
	if (err != hipSuccess) { std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n"; return -1; }

	// Starting threads to initialize the neighbour count grid
	initNeigh<<<dimGrid, dimBlock >>>(n, grid, neighGrid);
	err = hipGetLastError();
	if (err != hipSuccess) { std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }


	for (int a = 0; a < 10; ++a)
	{
		// Starting threads to step ahead in time 
		oneCell<<<dimGrid, dimBlock >>> (n, grid, neighGrid, neighGrid2);
		err = hipGetLastError();
		if (err != hipSuccess) { std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }
		
		// Copying the data back to the host
		err = hipMemcpy(v.data(), grid, n * n * sizeof(char), hipMemcpyDeviceToHost);
		if (err != hipSuccess) { std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

		printGrid(v, n);
	}

	// Copying the data back to the host
	err = hipMemcpy(v.data(), grid, n*n*sizeof(char), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

	// Freeing the device memory
	err = hipFree(neighGrid);
	if (err != hipSuccess) { std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree(neighGrid2);
	if (err != hipSuccess) { std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree(grid);
	if (err != hipSuccess) { std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

	printGrid(v, n);




	return 0;
}