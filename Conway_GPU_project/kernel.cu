﻿#include "cudaHelperFunctions.cuh"
#include "ConwayTable.hpp"
#include "time_meas.hpp"


int main()
{
	//Computational grid parameters
	
	

	//Host side variables
	
	//Beacon n = 6
	/*std::vector<char> v = {0,0,0,0,0,0,
							 0,1,1,0,0,0,
						     0,1,1,0,0,0,
						     0,0,0,1,1,0,
						     0,0,0,1,1,0,
						     0,0,0,0,0,0};
	int n = 6;
	dim3 dimBlock(6,6);*/

	//Blinker n = 5
	/*std::vector<char> v = {0,0,0,0,0,
						   0,0,1,0,0,
						   0,0,1,0,0,
						   0,0,1,0,0,
						   0,0,0,0,0};
	int n = 5;
	dim3 dimBlock(5, 5);*/


	// Glider
	std::vector<char> v = { 0,0,0,0,0,0,0,0,0,0,
							   0,0,1,0,0,0,0,0,0,0,
							   0,0,0,1,0,0,0,0,0,0,
							   0,1,1,1,0,0,0,0,0,0,
							   0,0,0,0,0,0,0,0,0,0,
							   0,0,0,0,0,0,0,0,0,0,
							   0,0,0,0,0,0,0,0,0,0,
							   0,0,0,0,0,0,0,0,0,0,
							   0,0,0,0,0,0,0,0,0,0,
							   0,0,0,0,0,0,0,0,0,0,
							   0,0,0,0,0,0,0,0,0,0 };
	int n = 20;
	dim3 dimGrid(1);
	dim3 dimBlock(20, 20);
	//int n = 1024;
	//std::vector<char> v = fillVector(0.5, n);
	
	
	
	//Device side variables
	char* grid = nullptr; //Conway table, from which the actual state of the cell is read

	char* grid2= nullptr; // A table where the new state of the cells is written
	

	// Allocating memory on the device 
	auto err = hipMalloc((void**)&grid, n * n * sizeof(char));
	if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
	err = hipMalloc((void**)&grid2, n * n * sizeof(char));
	if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
	

	// Copying data from the host to the device
	err = hipMemcpy(grid, v.data(), n * n * sizeof(char), hipMemcpyHostToDevice);
	if (err != hipSuccess) { std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n"; return -1; }

	

	std::ofstream f_output("cnw_gpu_visu.txt");

	std::cout << "Now, we startin'" << std::endl;
	for (int a = 0; a < 10; ++a)
	{
		auto t1 = tmark();

		// Starting threads to step ahead in time 
		oneCell<<<dimGrid, dimBlock>>> (n, grid, grid2);
		err = hipGetLastError();
		if (err != hipSuccess) { std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }
		
		// Copying the data back to the host
		err = hipMemcpy(v.data(), grid, n * n * sizeof(char), hipMemcpyDeviceToHost);
		if (err != hipSuccess) { std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

		// Handing the new neighGrid to the "read only" variable
		err = hipMemcpy(grid, grid2, n * n * sizeof(char), hipMemcpyDeviceToDevice);

		//printGrid(v, n);
		auto t2 = tmark();
		std::cout << delta_time(t1, t2) << std::endl;
		
	}
	
	
	// Copying the data back to the host
	err = hipMemcpy(v.data(), grid, n*n*sizeof(char), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

	// Freeing the device memory
	err = hipFree(grid);
	if (err != hipSuccess) { std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree(grid2);
	if (err != hipSuccess) { std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }


	return 0;
}